
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>

using namespace std;

#define max 250
#define TILE_SIZE 32

__host__ void init_matrix(int n, int32_t *matrix)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            matrix[(i * n) + j] = (rand() % max);
        }
    }
}

void printMat(int32_t *mat, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << mat[i * n + j] << " ";
        }
        cout << endl;
    }
    cout << endl;
}

__global__ void matmul(int32_t *A, int32_t *B, int32_t *C, int n)
{
    // Allocate shared memory
    __shared__ int32_t As[TILE_SIZE * TILE_SIZE];
    __shared__ int32_t Bs[TILE_SIZE * TILE_SIZE];

    // Compute row and column indices for tiles of C
    int row = (blockIdx.x * blockDim.x) + threadIdx.x;
    int col = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Initialize tile of C to 0
    int tile_sum = 0;

    // Loop over tiles of A and B that contribute to this tile of C
    for (int t = 0; t < n / TILE_SIZE; t++)
    {
        // Load tiles of A and B into shared memory
        As[threadIdx.x * TILE_SIZE + threadIdx.y] = A[row * n + t * TILE_SIZE + threadIdx.y];
        Bs[threadIdx.x * TILE_SIZE + threadIdx.y] = B[(t * TILE_SIZE + threadIdx.x) * n + col];
        __syncthreads();

        // Compute tile of C using tiles of A and B in shared memory
        for (int k = 0; k < TILE_SIZE; k++)
        {
            tile_sum += As[threadIdx.x * TILE_SIZE + k] * Bs[k * TILE_SIZE + threadIdx.y];
        }
        __syncthreads();
    }

    // Write tile of C to global memory
    C[row * n + col] = tile_sum;
}

__host__ int validate(int32_t *A, int32_t *B, int32_t *C, int n)
{
    int errors = 0;
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            int32_t C_check = 0;
            for (int k = 0; k < n; k++)
            {
                C_check += A[i * n + k] * B[k * n + j];
            }
            if (C_check != C[i * n + j])
            {
                errors += 1;
            }
        }
    }
    return errors;
}

int main(int argc, char **argv)
{
    hipError_t err;
    if (argc < 2)
    {
        cout << "usage: ./a.out <size_of_matrix>";
        return 0;
    }
    int n = atoi(argv[1]);
    int32_t *A = (int32_t *)malloc(n * n * sizeof(int32_t));
    int32_t *B = (int32_t *)malloc(n * n * sizeof(int32_t));
    int32_t *C = (int32_t *)malloc(n * n * sizeof(int32_t));
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            C[i * n + j] = 0;
        }
    }
    init_matrix(n, A);
    init_matrix(n, B);

    float data_transfer_time1, data_transfer_time2, computation_time;
    // some events to calculate execution time
    hipEvent_t start, stop;
    err = hipEventCreate(&start);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipEventCreate(&stop);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }

    // Allocating memory space on the device
    int32_t *d_A, *d_B, *d_C;
    err = hipMalloc((void **)&d_A, n * n * sizeof(int32_t));
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipMalloc((void **)&d_B, n * n * sizeof(int32_t));
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipMalloc((void **)&d_C, n * n * sizeof(int32_t));
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }

    // Recording time taken to transfer data from host to device
    err = hipEventRecord(start, 0);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }

    // Copying matrices A,B,C from host to device memory
    err = hipMemcpy(d_A, A, n * n * sizeof(int32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipMemcpy(d_B, B, n * n * sizeof(int32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    // cudaMemcpy(d_C, C, sizeof(int32_t)*n*n, cudaMemcpyHostToDevice);

    err = hipEventRecord(stop, 0);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipEventSynchronize(stop);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipEventElapsedTime(&data_transfer_time1, start, stop);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    // Recording time taken to compute matrix multiplication
    err = hipEventRecord(start, 0);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    cout << "Starting Matrix Multiplication..." << endl;
    dim3 numBlocks((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    matmul<<<numBlocks, blockSize>>>(d_A, d_B, d_C, n);
    err = hipEventRecord(stop, 0);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipEventSynchronize(stop);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipEventElapsedTime(&computation_time, start, stop);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }

    // Recording time taken to transfer result from device to host memory
    err = hipEventRecord(start, 0);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipMemcpy(C, d_C, n * n * sizeof(int32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipEventRecord(stop, 0);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipEventElapsedTime(&data_transfer_time2, start, stop);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }

    int mistakes = validate(A, B, C, n);
    if (mistakes != 0)
    {
        printf("Matrix multiplication was not correct, %d mistakes were reported\n", mistakes);
    }
    else
    {
        printf("Matrix multiplication was successfull!\n");
    }

    printf("Time taken to transfer data: %f\n", data_transfer_time1 + data_transfer_time2);
    printf("Time taken to perform the computation: %f\n", computation_time);
    // printMat(A,n);
    // printMat(B,n);
    // printMat(C,n);
    free(A);
    free(B);
    free(C);
    err = hipFree(d_A);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }
    err = hipFree(d_C);
    if (err != hipSuccess)
    {
        cout << hipGetErrorString(err) << endl;
        exit(1);
    }

    return 0;
}